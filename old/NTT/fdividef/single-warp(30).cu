#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <sstream>
using std::cout;
using std::endl;

#include "helper.h"
#include "parameter.h"

#define N 1024 * 4
#define nttBlockSize 1024
#define blockSize 256
#define blockCount N / blockSize
#define check 1

int size_array = sizeof(unsigned) * N;
int size = sizeof(unsigned);

__constant__ unsigned q_;
__constant__ unsigned ninv_;

__constant__ unsigned mu_;
__constant__ int bit_length_;

__global__ void barrett(unsigned a[], unsigned b[])
{

    register int i = blockIdx.x * blockSize + threadIdx.x;
    register int bit_length = bit_length_;

    register unsigned ra = a[i];
    register unsigned rb = b[i];

    unsigned long long rc, rx;

    rc = (unsigned long long)ra * rb;

    rx = rc >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    rc -= rx;

    if (rc < q_)
    {
        a[i] = rc;
    }
    else
    {
        a[i] = rc - q_;
    }
    
}

__global__ void inverseNTTbarrett(unsigned a[])
{
    register int i = blockIdx.x * blockSize + threadIdx.x;
    register int bit_length = bit_length_;

    register unsigned ra = a[i];
    unsigned long long rc, rx;

    rc = (unsigned long long)ra * ninv_;

    rx = rc >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    rc -= rx;

    if (rc < q_)
    {
        a[i] = rc;
    }
    else
    {
        a[i] = rc - q_;
    }

}

__device__ __forceinline__ void singleBarrett(unsigned long long& a)
{
    register int bit_length = bit_length_;

    unsigned long long rx;

    rx = a >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    a -= rx;

    if (a >= q_)
        a -= q_;
 
}

__global__ void CTBasedNTTInner(unsigned a[], unsigned psi_powers[])
{
    register int local_tid = threadIdx.x;

    register int z = psi_powers[local_tid % 32];

    #pragma unroll
    for (int length = 1; length < 17; length *= 2)
    {
        register int step = __fdividef(__fdividef(N, length), 2);

        #pragma unroll
        for (int iteration_num = 0; iteration_num < __fdividef(__fdividef(N, nttBlockSize), 2); iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = __fdividef(global_tid, step);
            register int target_index = psi_step * step * 2 + global_tid % step;

            //register unsigned psi = __shfl_sync(0xffffffff, z, (1 << length) - 1 + (global_tid / (N / (1 << length))));
            register unsigned psi = __shfl_sync(0xffffffff, z, length + psi_step);

            //register unsigned psi = psi_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned long long temp_storage = a[target_index + step];  // this is for eliminating the possibility of overflow

            temp_storage *= psi;

            singleBarrett(temp_storage);
            register unsigned second_target_value = temp_storage;

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            a[target_index + step] = first_target_value - second_target_value;
        }

        __syncthreads();
    }

    #pragma unroll
    for (int length = 32; length < N; length *= 2)
    {
        register int step = __fdividef(__fdividef(N, length), 2);

        #pragma unroll
        for (int iteration_num = 0; iteration_num < __fdividef(__fdividef(N, nttBlockSize), 2); iteration_num++)
        {

            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = __fdividef(global_tid, step);
            register int target_index = psi_step * step * 2 + global_tid % step;

            register unsigned psi = psi_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned long long temp_storage = a[target_index + step];  // this is for eliminating the possibility of overflow

            temp_storage *= psi;

            singleBarrett(temp_storage);
            register unsigned second_target_value = temp_storage;

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            a[target_index + step] = first_target_value - second_target_value;
        }

        __syncthreads();
    }

}

/*void CTBasedNTTMerged(unsigned long long a[], unsigned long long psi[], hipStream_t stream)
{
    #pragma unroll
    for (int m = 1; m < N; m *= 2)
    {
        CTBasedNTTInner << <N / nttBlockSize / 2, nttBlockSize, 1, stream >> > (a, psi, m);
    }
}*/

__global__ void GSBasedINTTInner(unsigned a[], unsigned psiinv_powers[])
{
    register int local_tid = threadIdx.x;

    register int z = psiinv_powers[local_tid % 32];

    #pragma unroll
    for (int length = __fdividef(N, 2); length > 16; length = __fdividef(length, 2))
    {
        register int step = __fdividef(__fdividef(N, length), 2);

        #pragma unroll
        for (int iteration_num = 0; iteration_num < __fdividef(__fdividef(N, nttBlockSize), 2); iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = __fdividef(global_tid, step);
            register int target_index = psi_step * step * 2 + global_tid % step;

            register unsigned psiinv = psiinv_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned second_target_value = a[target_index + step];

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            register unsigned long long temp_storage = first_target_value - second_target_value;

            temp_storage *= psiinv;

            singleBarrett(temp_storage);
            a[target_index + step] = temp_storage;
        }

        __syncthreads();
    }
    #pragma unroll
    for (int length = 16; length > 0; length = __fdividef(length, 2))
    {
        register int step = __fdividef(__fdividef(N, length), 2);

        #pragma unroll
        for (int iteration_num = 0; iteration_num < __fdividef(__fdividef(N, nttBlockSize), 2); iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = __fdividef(global_tid, step);
            register int target_index = psi_step * step * 2 + global_tid % step;

            //register unsigned psiinv = psiinv_powers[length + psi_step];
            register unsigned psiinv = __shfl_sync(0xffffffff, z, length + psi_step);

            register unsigned first_target_value = a[target_index];
            register unsigned second_target_value = a[target_index + step];

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            register unsigned long long temp_storage = first_target_value - second_target_value;

            temp_storage *= psiinv;

            singleBarrett(temp_storage);
            a[target_index + step] = temp_storage;
        }

        __syncthreads();
    }
}

void printArray(unsigned a[])
{
    std::string sss = "[";
    cout << sss;

    for (int i = 0; i < (N - 1); i++)
    {

        cout << a[i] << ", ";
    }

    cout << a[N - 1];

    cout << "]\n\n" << endl;
}

int main()
{
    hipSetDevice(2);

    unsigned q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams30(q, psi, psiinv, ninv, q_bit, N);

    unsigned psiTable[N];
    unsigned psiinvTable[N];
    fillTablePsi64(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi

    //copy powers of psi and psi inverse tables to device
    unsigned* d_psi;
    unsigned* d_psiinv;
    hipMalloc(&d_psi, size_array);
    hipMalloc(&d_psiinv, size_array);

    hipMemcpy(d_psi, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(d_psiinv, psiinvTable, size_array, hipMemcpyHostToDevice);

    cout << "n = " << N << endl;
    cout << "q = " << q << endl;
    cout << "Psi = " << psi << endl;
    cout << "Psi Inverse = " << psiinv << endl;

    //copy q and n inverse to gpu
    hipMemcpyToSymbol(HIP_SYMBOL(q_), &q, size);
    hipMemcpyToSymbol(HIP_SYMBOL(ninv_), &ninv, size);

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    unsigned long long mu1 = powl(2, 2 * bit_length);
    mu1 = mu1 / q;
    unsigned mu = mu1;

    //copy barrett parameters to device
    hipMemcpyToSymbol(HIP_SYMBOL(mu_), &mu, size);
    hipMemcpyToSymbol(HIP_SYMBOL(bit_length_), &bit_length, size);

    hipDeviceSynchronize();

    unsigned* a = (unsigned*)malloc(sizeof(unsigned) * N);
    randomArray64(a, N, q - 1); //fill array with random numbers between 0 and q - 1

    unsigned* b = (unsigned*)malloc(sizeof(unsigned) * N);
    randomArray64(b, N, q - 1); //fill array with random numbers between 0 and q - 1

    unsigned* refc;
    if (check)
        refc = refPolyMul64(a, b, q, N);

    hipDeviceSynchronize();

    float total_time = 0;

    float time;
    hipEvent_t startf, stopf;

    hipEventCreate(&startf);
    hipEventCreate(&stopf);
    hipEventRecord(startf, 0);

    //copy random arrays to gpu
    unsigned* d_a;
    hipMalloc(&d_a, size_array);
    hipMemcpy(d_a, a, size_array, hipMemcpyHostToDevice);
    unsigned* d_b;
    hipMalloc(&d_b, size_array);
    hipMemcpy(d_b, b, size_array, hipMemcpyHostToDevice);

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    printf("\nTime taken to copy input polynomials to device:  %f ms \n", time);
    total_time += time;

    hipDeviceSynchronize();

    hipStream_t ntt1, ntt2;
    hipStreamCreate(&ntt1);
    hipStreamCreate(&ntt2);

    hipEventRecord(startf, 0);

    CTBasedNTTInner << <1, nttBlockSize, 0, 0 >> > (d_a, d_psi);

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    printf("\nTime taken to perform 1st forward NTT:  %f ms \n", time);
    total_time += time;

    hipEventRecord(startf, 0);

    CTBasedNTTInner << <1, nttBlockSize, 0, 0 >> > (d_b, d_psi);

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);

    printf("\nTime taken to perform 2nd forward NTT:  %f ms \n", time);
    total_time += time;

    hipEventRecord(startf, 0);

    barrett<<<blockCount, blockSize>>>(d_a, d_b);    

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    printf("\nTime taken to perform coefficient multiplication with barrett:  %f ms \n", time);
    total_time += time;

    hipEventRecord(startf, 0);

    GSBasedINTTInner << <1, nttBlockSize>> > (d_a, d_psiinv);

    inverseNTTbarrett << <blockCount, blockSize>> > (d_a);

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    printf("\nTime taken to perform inverse NTT:  %f ms \n", time);
    total_time += time;

    hipEventRecord(startf, 0);
    
    hipMemcpy(a, d_a, size_array, hipMemcpyDeviceToHost);    

    hipEventRecord(stopf, 0);
    hipEventSynchronize(stopf);
    hipEventElapsedTime(&time, startf, stopf);

    hipDeviceSynchronize();

    hipEventDestroy(startf); hipEventDestroy(stopf);

    printf("\nTime taken to copy results back to host:  %f ms \n", time);
    total_time += time;

    printf("\nTotal execution time:  %f ms \n", total_time);
    
    if (check) //check the correctness of results
    {
        for (int i = 0; i < N; i++)
        {
            if (a[i] != refc[i])
            {
                //cout << "error" << endl;
                cout << "i:   " << i << ", Computed result:   " << a[i] << ", Reference result:   " << refc[i] << endl;
            }
                
        }

        free(refc);
    }

    free(a); free(b);

    return 0;
}
