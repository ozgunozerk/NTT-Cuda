#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <sstream>
using std::cout;
using std::endl;

#include "helper.h"
#include "parameter.h"

#define N 1024 * 64
#define nttBlockSize 1024
#define blockSize 256
#define blockCount N / blockSize
#define check 0

int size_array = sizeof(unsigned) * N;
int size = sizeof(unsigned);

__constant__ unsigned q_;
__constant__ unsigned ninv_;

__constant__ unsigned mu_;
__constant__ int bit_length_;

__global__ void barrett(unsigned a[], unsigned b[])
{

    register int i = blockIdx.x * blockSize + threadIdx.x;
    register int bit_length = bit_length_;

    register unsigned ra = a[i];
    register unsigned rb = b[i];

    unsigned long long rc, rx;

    rc = (unsigned long long)ra * rb;

    rx = rc >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    rc -= rx;

    if (rc < q_)
    {
        a[i] = rc;
    }
    else
    {
        a[i] = rc - q_;
    }
    
}

__global__ void inverseNTTbarrett(unsigned a[])
{
    register int i = blockIdx.x * blockSize + threadIdx.x;
    register int bit_length = bit_length_;

    register unsigned ra = a[i];
    unsigned long long rc, rx;

    rc = (unsigned long long)ra * ninv_;

    rx = rc >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    rc -= rx;

    if (rc < q_)
    {
        a[i] = rc;
    }
    else
    {
        a[i] = rc - q_;
    }

}

__device__ __forceinline__ void singleBarrett(unsigned long long& a)
{
    register int bit_length = bit_length_;

    unsigned long long rx;

    rx = a >> (bit_length - 2);

    rx *= mu_;

    rx >>= (bit_length + 2);

    rx *= q_;

    a -= rx;

    if (a >= q_)
        a -= q_;
 
}

__global__ void CTBasedNTTInner(unsigned a[], unsigned psi_powers[])
{
    register int local_tid = threadIdx.x;

    register int z = psi_powers[local_tid % 32];

    #pragma unroll
    for (int length = 1; length < 17; length *= 2)
    {
        register int step = (N / length) / 2;


        for (int iteration_num = 0; iteration_num < (N / nttBlockSize) / 2; iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = global_tid / step;
            register int target_index = psi_step * step * 2 + global_tid % step;

            //register unsigned psi = __shfl_sync(0xffffffff, z, (1 << length) - 1 + (global_tid / (N / (1 << length))));
            register unsigned psi = __shfl_sync(0xffffffff, z, length + psi_step);

            //register unsigned psi = psi_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned long long temp_storage = a[target_index + step];  // this is for eliminating the possibility of overflow

            temp_storage *= psi;

            singleBarrett(temp_storage);
            register unsigned second_target_value = temp_storage;

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            a[target_index + step] = first_target_value - second_target_value;
        }

        __syncthreads();
    }

    #pragma unroll
    for (int length = 32; length < N; length *= 2)
    {
        register int step = (N / length) / 2;


        for (int iteration_num = 0; iteration_num < (N / nttBlockSize) / 2; iteration_num++)
        {

            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = global_tid / step;
            register int target_index = psi_step * step * 2 + global_tid % step;

            register unsigned psi = psi_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned long long temp_storage = a[target_index + step];  // this is for eliminating the possibility of overflow

            temp_storage *= psi;

            singleBarrett(temp_storage);
            register unsigned second_target_value = temp_storage;

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            a[target_index + step] = first_target_value - second_target_value;
        }

        __syncthreads();
    }

}

/*void CTBasedNTTMerged(unsigned long long a[], unsigned long long psi[], hipStream_t stream)
{
    #pragma unroll
    for (int m = 1; m < N; m *= 2)
    {
        CTBasedNTTInner << <N / nttBlockSize / 2, nttBlockSize, 1, stream >> > (a, psi, m);
    }
}*/

__global__ void GSBasedINTTInner(unsigned a[], unsigned psiinv_powers[])
{
    register int local_tid = threadIdx.x;

    register int z = psiinv_powers[local_tid % 32];

    #pragma unroll
    for (int length = (N / 2); length > 16; length /= 2)
    {
        register int step = (N / length) / 2;


        for (int iteration_num = 0; iteration_num < (N / nttBlockSize) / 2; iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = global_tid / step;
            register int target_index = psi_step * step * 2 + global_tid % step;

            register unsigned psiinv = psiinv_powers[length + psi_step];

            register unsigned first_target_value = a[target_index];
            register unsigned second_target_value = a[target_index + step];

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            register unsigned long long temp_storage = first_target_value - second_target_value;

            temp_storage *= psiinv;

            singleBarrett(temp_storage);
            a[target_index + step] = temp_storage;
        }

        __syncthreads();
    }
    #pragma unroll
    for (int length = 16; length > 0; length /= 2)
    {
        register int step = (N / length) / 2;


        for (int iteration_num = 0; iteration_num < (N / nttBlockSize) / 2; iteration_num++)
        {
            register int global_tid = local_tid + iteration_num * nttBlockSize;

            register int psi_step = global_tid / step;
            register int target_index = psi_step * step * 2 + global_tid % step;

            //register unsigned psiinv = psiinv_powers[length + psi_step];
            register unsigned psiinv = __shfl_sync(0xffffffff, z, length + psi_step);

            register unsigned first_target_value = a[target_index];
            register unsigned second_target_value = a[target_index + step];

            register unsigned target_result = first_target_value + second_target_value;

            if (target_result >= q_)
                target_result -= q_;

            a[target_index] = target_result;

            if (first_target_value < second_target_value)
                first_target_value += q_;

            register unsigned long long temp_storage = first_target_value - second_target_value;

            temp_storage *= psiinv;

            singleBarrett(temp_storage);
            a[target_index + step] = temp_storage;
        }

        __syncthreads();
    }
}

void printArray(unsigned a[])
{
    std::string sss = "[";
    cout << sss;

    for (int i = 0; i < (N - 1); i++)
    {

        cout << a[i] << ", ";
    }

    cout << a[N - 1];

    cout << "]\n\n" << endl;
}

int main()
{
    hipSetDevice(2);

    unsigned q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams30(q, psi, psiinv, ninv, q_bit, N);

    unsigned psiTable[N];
    unsigned psiinvTable[N];
    fillTablePsi64(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi

    //copy powers of psi and psi inverse tables to device
    unsigned* d_psi;
    unsigned* d_psiinv;
    hipMalloc(&d_psi, size_array);
    hipMalloc(&d_psiinv, size_array);

    hipMemcpy(d_psi, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(d_psiinv, psiinvTable, size_array, hipMemcpyHostToDevice);

    cout << "n = " << N << endl;
    cout << "q = " << q << endl;
    cout << "Psi = " << psi << endl;
    cout << "Psi Inverse = " << psiinv << endl;

    //copy q and n inverse to gpu
    hipMemcpyToSymbol(HIP_SYMBOL(q_), &q, size);
    hipMemcpyToSymbol(HIP_SYMBOL(ninv_), &ninv, size);

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    unsigned long long mu1 = powl(2, 2 * bit_length);
    mu1 = mu1 / q;
    unsigned mu = mu1;

    //copy barrett parameters to device
    hipMemcpyToSymbol(HIP_SYMBOL(mu_), &mu, size);
    hipMemcpyToSymbol(HIP_SYMBOL(bit_length_), &bit_length, size);

    unsigned* a;
    hipHostMalloc(&a, sizeof(unsigned) * N);
    randomArray64(a, N, q - 1); //fill array with random numbers between 0 and q - 1

    unsigned* b;
    hipHostMalloc(&b, sizeof(unsigned) * N);
    randomArray64(b, N, q - 1); //fill array with random numbers between 0 and q - 1

    unsigned* d_a;
    hipMalloc(&d_a, size_array);
    unsigned* d_b;
    hipMalloc(&d_b, size_array);

    unsigned* refc;
    if (check)
        refc = refPolyMul64(a, b, q, N);

    hipStream_t ntt1, ntt2;
    hipStreamCreate(&ntt1);
    hipStreamCreate(&ntt2);

    //copy random arrays to gpu
    
    hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
    hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);

    CTBasedNTTInner << <1, nttBlockSize, 0, ntt1 >> > (d_a, d_psi);
    CTBasedNTTInner << <1, nttBlockSize, 0, ntt2 >> > (d_b, d_psi);

    barrett<<<blockCount, blockSize, 0, 0>>>(d_a, d_b);    

    GSBasedINTTInner << <1, nttBlockSize, 0, 0>> > (d_a, d_psiinv);
 
    inverseNTTbarrett << <blockCount, blockSize, 0, 0>> > (d_a);
    
    hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, ntt1);    

    hipDeviceSynchronize();

    hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);
    
    if (check) //check the correctness of results
    {
        for (int i = 0; i < N; i++)
        {
            if (a[i] != refc[i])
            {
                cout << "error" << endl;
                cout << i << "   " << a[i] << "   " << refc[i] << endl;
            }
                
        }

        free(refc);
    }

    hipHostFree(a); hipHostFree(b);

    return 0;
}
