#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"
#include "distributions.cuh"

#define check 1

int main1()
{
    int n = 1024 * 4;

    vector<unsigned long long> q = { 68719403009, 68719230977, 137438822401 };
    //vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };
    vector<unsigned> q_bit_lengths = { 36, 36, 37 };
    unsigned q_amount = q.size();

    // run operations on different q's with different streams
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount);
    for (int i = 0; i < q_amount; i++)
        hipStreamCreate(&streams[i]);
    
    unsigned char* in;
    hipMalloc(&in, (sizeof(char) + sizeof(unsigned long long)) * q_amount * n + sizeof(unsigned) * n);

    unsigned long long** secret_key = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&secret_key[i], sizeof(unsigned long long) * n);
    }
    
    // we always have 2 public keys
    unsigned long long*** public_key = (unsigned long long***)malloc(sizeof(unsigned long long**) * 2);
    public_key[0] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    public_key[1] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            hipMalloc(&public_key[i][j], sizeof(unsigned long long) * n);
        }
    }

    generate_random(in, (sizeof(char) + sizeof(unsigned long long)) * q_amount * n + sizeof(unsigned) * n, streams[0]);

    // convert random bytes to ternary distribution
    // use different byte sequences for each element of the secret key
    for (int i = 0; i < q_amount; i++)
    {
        ternary_dist(in + i * n, secret_key[i], n, streams[i], q[i]);
    }

    // convert random bytes to uniform distribution
    // use different byte sequences for each q
    for (int i = 0; i < q_amount; i++)
    {
        uniform_dist((unsigned long long*)(in + q_amount * n + i * n * sizeof(unsigned long long)), public_key[1][i], n, streams[i], q[i]);
    }

    // a temp array to store gaussian distribution values (e)
    unsigned long long** temp = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&temp[i], sizeof(unsigned long long) * q_amount * n);
    }

    for (int i = 0; i < q_amount; i++)
    {
        gaussian_dist((unsigned*)(in + q_amount * n + q_amount * n * sizeof(unsigned long long)), temp[i], n, streams[i], q[i]);
    }

    vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };
    vector<unsigned long long> psiinv_roots = { 60243494989, 37410665880, 5716440802 };
    vector<unsigned long long> mu_array = {};

    //generate mu parameters for barrett
    for (int i = 0; i < q_amount; i++)
    {
        unsigned int bit_length = q_bit_lengths[i];
        uint128_t mu1 = uint128_t::exp2(2 * bit_length);
        mu1 = mu1 / q[i];
        unsigned long long mu = mu1.low;

        mu_array.push_back(mu);
    }

    //allocate memory for powers of psi root and psi inverse root
    //and fill those arrays
    unsigned long long** psi_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        psi_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
        psiinv_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

        fillTablePsi128(psi_roots[i], q[i], psiinv_roots[i], psi_table[i], psiinv_table[i], n);
    }

    //allocate memory for powers of psi root and psi inverse root on device
    //and copy their values from host to device
    unsigned long long** psi_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&psi_table_device[i], sizeof(unsigned long long) * n);
        hipMalloc(&psiinv_table_device[i], sizeof(unsigned long long) * n);

        hipMemcpy(psi_table_device[i], psi_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
        hipMemcpy(psiinv_table_device[i], psiinv_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
    }

    for (int i = 0; i < q_amount; i++)
    {
        hipMemcpyAsync(public_key[0][i], public_key[1][i], sizeof(unsigned long long) * n, hipMemcpyDeviceToDevice, streams[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        half_poly_mul_device(public_key[0][i], secret_key[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device[i], psiinv_table_device[i]);
        poly_add_device(public_key[0][i], secret_key[i], n, streams[i], q[i]);
        poly_negate_device(public_key[0][i], n, streams[i], q[i]);
    }

    return 0;
}

int main()
{
    //unsigned N = atoi(argv[1]);
    unsigned N = 1024 * 4;

    int size_array = sizeof(unsigned long long) * N;
    int size = sizeof(unsigned long long);

    unsigned long long q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams(q, psi, psiinv, ninv, q_bit, N);

    unsigned long long* psiTable = (unsigned long long*)malloc(size_array);
    unsigned long long* psiinvTable = (unsigned long long*)malloc(size_array);
    fillTablePsi128(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi

    //copy powers of psi and psi inverse tables to device
    unsigned long long* psi_powers, * psiinv_powers;

    hipMalloc(&psi_powers, size_array);
    hipMalloc(&psiinv_powers, size_array);

    hipMemcpy(psi_powers, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(psiinv_powers, psiinvTable, size_array, hipMemcpyHostToDevice);

    cout << "n = " << N << endl;
    cout << "q = " << q << endl;
    cout << "Psi = " << psi << endl;
    cout << "Psi Inverse = " << psiinv << endl;

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    uint128_t mu1 = uint128_t::exp2(2 * bit_length);
    mu1 = mu1 / q;

    unsigned long long mu = mu1.low;

    unsigned long long* a;
    hipHostMalloc(&a, sizeof(unsigned long long) * N);
    randomArray128(a, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* b;
    hipHostMalloc(&b, sizeof(unsigned long long) * N);
    randomArray128(b, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* d_a;
    hipMalloc(&d_a, size_array);
    unsigned long long* d_b;
    hipMalloc(&d_b, size_array);

    unsigned long long* refc;
    if (check)
        refc = refPolyMul128(a, b, q, N);

    hipStream_t ntt1, ntt2;
    hipStreamCreate(&ntt1);
    hipStreamCreate(&ntt2);

    /*hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
    hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);

    full_poly_mul_device(d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers, psiinv_powers);

    hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, 0);*/

    unsigned long long* result = full_poly_mul(a, b, d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers, psiinv_powers);

    hipDeviceSynchronize();

    hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);

    if (check) //check the correctness of results
    {
        for (int i = 0; i < N; i++)
        {
            if (result[i] != refc[i])
            {
                cout << "error" << endl;
                cout << i << "   " << a[i] << "   " << refc[i] << endl;
            }

        }

        free(refc);
    }

    hipHostFree(a); hipHostFree(b);

    return 0;
}

/*int main()
{
    int n = 1024 * 4;

    unsigned long long q = 68719403009;

    unsigned char* in;
    unsigned long long* out;
    unsigned long long* host;

    hipHostMalloc(&host, n * sizeof(unsigned long long));

    hipMalloc(&in, n * sizeof(char)); hipMalloc(&out, n * sizeof(unsigned long long));

    hipStream_t stream; hipStreamCreate(&stream);

    generate_random(in, n, stream, 1);
    uniform_dist((unsigned long long*)in, out, n, stream, q);

    hipMemcpyAsync(host, out, n * sizeof(unsigned long long), hipMemcpyDeviceToHost, stream);

    hipDeviceSynchronize();

    for (int i = 0; i < n; i++)
    {
        printf("%llu\n", host[i]);
    }

    return 0;
}*/


