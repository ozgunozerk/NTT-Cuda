#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"
#include "distributions.cuh"

void divide_and_round_q_last_inplace(unsigned long long* poly, unsigned N, hipStream_t streams[], vector<unsigned long long> q, vector<unsigned> q_bit_lengths, 
    vector<unsigned long long> mu_array, vector<unsigned long long> inv_q_last_mod_q)
{
    unsigned q_amount = q.size();  // getting how many q's we have

    unsigned long long last_modulus = q[q_amount - 1];  // get the last q from the array
    unsigned long long half_last_modulus = last_modulus >> 1;  // divide it by half

    poly_add_integer_device_default(poly + N * (q_amount - 1), half_last_modulus, N, last_modulus);  
    // poly + N * (q_amount - 1) = getting the to the last q in the flattened array
    // adding half_last_modulus to it, in mod last_modulus
    // N is required for calling the kernel with optimal thread amount

    for (int i = 0; i < q_amount - 1; i++)  
    {
        unsigned long long half_mod = half_last_modulus % q[i];  // getting the half_last_modulus' mod in respect to every other q[x]
        divide_and_round_q_last_inplace_loop << <N / 256, 256, 0, streams[i] >> > (poly + N * i, poly + N * (q_amount - 1), q[i], half_mod, inv_q_last_mod_q[i], mu_array[i], q_bit_lengths[i]);  // too long to explain, inspect the comments in the function
    }
}

void rns_encryption(unsigned long long* c0, unsigned long long* c1, unsigned long long*** public_key, unsigned char* in, unsigned long long** u, unsigned long long*** e, unsigned N,
    hipStream_t streams[], vector<unsigned long long> q, vector<unsigned> q_bit_lengths,
    vector<unsigned long long> mu_array, vector<unsigned long long> inv_q_last_mod_q, unsigned long long** psi_table_device, unsigned long long** psiinv_table_device,
    unsigned long long* m_poly_device, unsigned long long m_len, unsigned long long* qi_div_t_rns_array_device, unsigned long long* q_array_device, unsigned t)
{
    unsigned q_amount = q.size();  // getting how many q's do we have

    generate_random_default(in, sizeof(unsigned long long) * q_amount * N + sizeof(unsigned) * N * 2);  // default is for default stream: this is for synchronization issues
    // otherwise ternary distributions may run before this function, which is UNACCEPTABLE

    for (int i = 0; i < q_amount; i++)
    {
        ternary_dist(in + i * N, c0 + i * N, N, streams[i], q[i]);  // generate ternary dist poly directly into c0 and c1. c0 = c1, 
        ternary_dist(in + i * N, c1 + i * N, N, streams[i], q[i]);  // its represented by 'u' 
        // for ease of operations and memory allocation, we have generated 2 of them (identical), since we override some stuff in polynomial multiplication.
    }

    for (int i = 0; i < q_amount; i++)
    {
        gaussian_dist((unsigned*)(in + q_amount * N), e[0][i], N, streams[i], q[i]);  // this is again for generation ternary distribution, although it's name is gaussian
        // e0

        gaussian_dist((unsigned*)(in + q_amount * N), e[1][i], N, streams[i], q[i]);  // i was joking this is for gaussian
        // e1
    }

    // CAN should we delete the comment below?
    // olur
    /*c0 = self.pk[0] * u
        c1 = self.pk[1] * u
        c0 = c0 + e1
        c1 = c1 + e2*/

    for (int i = 0; i < q_amount; i++)
    {
        // multiply each public key with 'u'(c0 and c1). Remember that c0 and c1 are identical
        half_poly_mul_device(c0 + i * N, public_key[0][i], N, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device[i], psiinv_table_device[i]);  
        half_poly_mul_device(c1 + i * N, public_key[1][i], N, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device[i], psiinv_table_device[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        poly_add_device(c0 + i * N, e[0][i], N, streams[i], q[i]);  // add e0 to publickey[0]
        poly_add_device(c1 + i * N, e[1][i], N, streams[i], q[i]);  // add e1 to publickey[1]
    }

    divide_and_round_q_last_inplace(c0, N, streams, q, q_bit_lengths, mu_array, inv_q_last_mod_q);  // do that complicated stuff for each public key
    divide_and_round_q_last_inplace(c1, N, streams, q, q_bit_lengths, mu_array, inv_q_last_mod_q);

    weird_m_stuff << <N / 256, 256, 0, 0 >> > (m_len, m_poly_device, c0, t, qi_div_t_rns_array_device, q_array_device, q_amount, N);  // look at the comments in the function
}

int main()
{
    int n = 1024 * 4;

    vector<unsigned long long> q = { 68719403009, 68719230977, 137438822401 };  // the first value is 68719403009 because its bigger than 68719403008
    unsigned long long q_array[] = { 68719403009, 68719230977, 137438822401 };
    vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };
    vector<unsigned long long> psiinv_roots = { 60243494989, 37410665880, 5716440802 };
    vector<unsigned> q_bit_lengths = { 36, 36, 37 };
    vector<unsigned long long> mu_array = {};
    unsigned q_amount = q.size();
    vector<unsigned long long> inv_q_last_mod_q = { 20955999355, 17095778744 };
    unsigned long long q_mod_t = 1;
    vector<unsigned long long> qi_div_t_rns = { 67108792, 67108624, 134217600 }; // we don't need this vector
    unsigned long long qi_div_t_rns_array[] = { 67108792, 67108624, 134217600 };

    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount);
    for (int i = 0; i < q_amount; i++)
        hipStreamCreate(&streams[i]);  // create streams for parallelism

    unsigned long long* q_array_device;
    hipMalloc(&q_array_device, q_amount * sizeof(unsigned long long));
    hipMemcpy(q_array_device, q_array, q_amount * sizeof(unsigned long long), hipMemcpyHostToDevice);

    unsigned long long* qi_div_t_rns_array_device;
    hipMalloc(&qi_div_t_rns_array_device, (q_amount - 1)* sizeof(unsigned long long));
    hipMemcpy(qi_div_t_rns_array_device, qi_div_t_rns_array, (q_amount - 1) * sizeof(unsigned long long), hipMemcpyHostToDevice);

    unsigned long long m = 100;  // our message to encrypt
    unsigned long long m_len = log2(m) + 1;  // length of m

    unsigned long long t = 1024;  // mathematical stuff that is beyond our comprehension

    //generate mu parameters for barrett
    for (int i = 0; i < q_amount; i++)
    {
        unsigned int bit_length = q_bit_lengths[i];
        uint128_t mu1 = uint128_t::exp2(2 * bit_length);
        mu1 = mu1 / q[i];
        unsigned long long mu = mu1.low;

        mu_array.push_back(mu);
    }

    //allocate memory for powers of psi root and psi inverse root
    //and fill those arrays
    unsigned long long** psi_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        psi_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
        psiinv_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

        fillTablePsi128(psi_roots[i], q[i], psiinv_roots[i], psi_table[i], psiinv_table[i], n);
    }

    //allocate memory for powers of psi root and psi inverse root on device
    //and copy their values from host to device
    unsigned long long** psi_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&psi_table_device[i], sizeof(unsigned long long) * n);
        hipMalloc(&psiinv_table_device[i], sizeof(unsigned long long) * n);

        hipMemcpy(psi_table_device[i], psi_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
        hipMemcpy(psiinv_table_device[i], psiinv_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
    }

    unsigned long long* c0;
    unsigned long long* c1;
    hipMalloc(&c0, sizeof(unsigned long long) * n * q_amount);
    hipMalloc(&c1, sizeof(unsigned long long) * n * q_amount);

    unsigned char* in;
    hipMalloc(&in, sizeof(unsigned long long) * q_amount * n + sizeof(unsigned) * n * 2);

    unsigned long long** u = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&u[i], sizeof(unsigned long long) * n);
    }

    unsigned long long*** e = (unsigned long long***)malloc(sizeof(unsigned long long**) * 2);
    e[0] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    e[1] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            hipMalloc(&e[i][j], sizeof(unsigned long long) * n);
        }
    }

    unsigned long long*** public_key = (unsigned long long***)malloc(sizeof(unsigned long long**) * 2);
    public_key[0] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    public_key[1] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);

    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            hipMalloc(&public_key[i][j], sizeof(unsigned long long) * n);
        }
    }

    unsigned long long m_poly[4096];
    for (int i = 0; i < 4096; i++)
    {
        m_poly[i] = 0;
    }

    m_poly[2] = 1; m_poly[5] = 1; m_poly[6] = 1;

    unsigned long long* m_poly_device;
    hipMalloc(&m_poly_device, 4096 * sizeof(unsigned long long));
    hipMemcpy(m_poly_device, m_poly, 4096 * sizeof(unsigned long long), hipMemcpyHostToDevice);

    rns_encryption(c0, c1, public_key, in, u, e, n, streams, q, q_bit_lengths, mu_array, 
        inv_q_last_mod_q, psi_table_device, psiinv_table_device, m_poly_device, m_len, qi_div_t_rns_array_device, q_array_device, t);

}