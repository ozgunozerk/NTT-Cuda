#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"
#include "distributions.cuh"
#include "bfv_keygen.cuh"
#include "bfv_encryption.cuh"
#include "bfv_decryption.cuh"

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float keygen = 0, enc = 0, dec = 0;

	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	int n = 1024 * 32;

	unsigned long long t = 1024;  // mathematical stuff that is beyond our comprehension

	// 32k 9q
	//unsigned long long q_array[] = { 36028797012606977, 36028797010444289, 36028797009985537, 36028797005856769, 36028797005529089, 36028797005135873, 36028797003694081, 36028797003563009, 36028797001138177 };
	//vector<unsigned long long> psi_roots = { 768741990072, 3911086673862, 5947090524825, 47595902954, 2691682578057, 3903338373, 235185854118, 1769787302793, 3151164484090 };

	// 32k 16q
	unsigned long long q_array[] = { 18014398506729473, 36028797017456641, 36028797014704129, 36028797014573057, 36028797014376449, 36028797013327873, 36028797013000193, 36028797012606977, 36028797010444289, 36028797009985537, 36028797005856769, 36028797005529089, 36028797005135873, 36028797003694081, 36028797003563009, 36028797001138177 };
	vector<unsigned long long> psi_roots = { 58232959302, 1155186985540, 631260524634, 1526647220035, 455957817523, 1650884166641, 10316746886, 768741990072, 3911086673862, 5947090524825, 47595902954, 2691682578057, 3903338373, 235185854118, 1769787302793, 3151164484090 };

	// 32k 11q
	//unsigned long long q_array[] = { 36028797013327873, 36028797013000193, 36028797012606977, 36028797010444289, 36028797009985537, 36028797005856769, 36028797005529089, 36028797005135873, 36028797003694081, 36028797003563009, 36028797001138177 };
	//vector<unsigned long long> psi_roots = { 1650884166641, 10316746886, 768741990072, 3911086673862, 5947090524825, 47595902954, 2691682578057, 3903338373, 235185854118, 1769787302793, 3151164484090 };

	// 8k 3q
	//unsigned long long q_array[] = { 274877562881, 274877202433, 274877153281 };
	//vector<unsigned long long> psi_roots = { 71485851, 33872056, 22399294 };

	// 4k 3q
	//unsigned long long q_array[] = { 68719403009, 68719230977, 137438822401 };
	//vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };

	// 8k 4q
	//unsigned long long q_array[] = { 8796092858369, 8796092792833, 17592186028033, 17592185438209 };
	//vector<unsigned long long> psi_roots = { 1734247217, 304486499, 331339694, 9366611238 };

	// 16k 5q
	//unsigned long long q_array[] = { 1125899904679937, 1125899903991809, 1125899903827969, 1125899903795201, 1125899903500289 };
	//vector<unsigned long long> psi_roots = { 184459094098, 125929543876, 13806300337, 10351677219, 68423600398 };

	//16k 9q
	//unsigned long long q_array[] = { 281474976546817, 281474976317441, 281474975662081, 562949952798721, 562949952700417, 562949952274433, 562949951979521, 562949951881217, 1125899904679937 };
	//vector<unsigned long long> psi_roots = { 23720796222, 21741529212, 13412349256, 1196930505, 31695302805, 6575376104, 394024808, 45092463253, 184459094098 };

	vector<unsigned long long> psiinv_roots;
	vector<unsigned long long> mu_array = {};
	unsigned q_amount = sizeof(q_array) / sizeof(unsigned long long);
	vector<unsigned> q_bit_lengths;
	unsigned* q_bit_lengths_p = (unsigned*)malloc(sizeof(unsigned) * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		q_bit_lengths.push_back(log2((double)q_array[i]) + 1);
		q_bit_lengths_p[i] = q_bit_lengths[i];
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(q_bit_cons), q_bit_lengths_p, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice, 0);
	vector<unsigned long long> inv_q_last_mod_q;
	unsigned long long* inv_q_last_mod_q_p = (unsigned long long*)malloc(sizeof(unsigned long long) * (q_amount - 1));
	for (int i = 0; i < q_amount - 1; i++)
	{
		inv_q_last_mod_q.push_back(modinv128(q_array[q_amount - 1] % q_array[i], q_array[i]));
		inv_q_last_mod_q_p[i] = inv_q_last_mod_q[i];
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(inv_q_last_mod_q_cons), inv_q_last_mod_q_p, sizeof(unsigned long long) * (q_amount - 1), 0, hipMemcpyHostToDevice, 0);
	unsigned long long q_mod_t = 1;
	//unsigned long long qi_div_t_rns_array[] = { 67108792, 67108624, 134217600 };
	unsigned long long* qi_div_t_rns_array = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		qi_div_t_rns_array[i] = q_array[i] / t;
	}
	vector<unsigned long long> punctured_q = {};
	//vector<unsigned long long> inv_punctured_q = { 26179219651, 42540076863 };
	vector<unsigned long long> inv_punctured_q;
	unsigned long long* inv_punctured_q_array = (unsigned long long*)malloc(sizeof(unsigned long long) * (q_amount - 1));
	unsigned long long gamma = 2305843009213683713;
	unsigned long long gamma_div_2 = gamma >> 1;

	for (int i = 0; i < q_amount; i++)
		psiinv_roots.push_back(modinv128(psi_roots[i], q_array[i]));

	vector<unsigned long long> output_base = { t, gamma };
	vector<unsigned> output_base_bit_lengths = { 10, 61 };
	unsigned long long mu_gamma;

	unsigned long long mult_t = 1, mult_g = 1;
	for (int i = 0; i < q_amount - 1; i++)
	{
		mult_t = (host64x2(mult_t, q_array[i]) % t).low;
		mult_g = (host64x2(mult_g, q_array[i]) % gamma).low;
	}
	mult_t = t - modinv128(mult_t, t);
	mult_g = gamma - modinv128(mult_g, gamma);

	vector<unsigned long long> neg_inv_qs_mult_mod_t_gamma = { mult_t, mult_g }; // all qs multiplied mod t and gamma then taken inverse then negated

	//vector<unsigned long long> prod_t_gamma_mod_q = { 37067052033, 64547873793 };
	vector<unsigned long long> prod_t_gamma_mod_q;
	unsigned long long* prod_t_gamma_mod_q_array = (unsigned long long*)malloc(sizeof(unsigned long long) * (q_amount - 1));

	uint128_t prod_t_gamma = host64x2(t, gamma);
	for (int i = 0; i < q_amount - 1; i++)
	{
		prod_t_gamma_mod_q.push_back((prod_t_gamma % q_array[i]).low);
		prod_t_gamma_mod_q_array[i] = prod_t_gamma_mod_q[i];
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(prod_t_gamma_mod_q_cons), prod_t_gamma_mod_q_array, sizeof(unsigned long long) * (q_amount - 1), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(q_cons), q_array, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice);

	// run operations on different q's with different streams
	hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount * 2);
	for (int i = 0; i < q_amount * 2; i++)
		hipStreamCreate(&streams[i]);

	unsigned long long* q_array_device;
	hipMalloc(&q_array_device, q_amount * sizeof(unsigned long long));
	hipMemcpy(q_array_device, q_array, q_amount * sizeof(unsigned long long), hipMemcpyHostToDevice);

	unsigned long long* qi_div_t_rns_array_device;
	hipMalloc(&qi_div_t_rns_array_device, q_amount * sizeof(unsigned long long));
	hipMemcpy(qi_div_t_rns_array_device, qi_div_t_rns_array, q_amount * sizeof(unsigned long long), hipMemcpyHostToDevice);

	unsigned char* in;
	hipMalloc(&in, (sizeof(char) + sizeof(unsigned long long)) * q_amount * n + sizeof(unsigned) * n);

	unsigned long long* secret_key;
	hipMalloc(&secret_key, sizeof(unsigned long long) * n * q_amount);

	// we always have 2 public keys
	unsigned long long* public_key;
	hipMalloc(&public_key, sizeof(unsigned long long) * 2 * n * q_amount);

	// a temp array to store gaussian distribution values (e)
	unsigned long long* temp;
	hipMalloc(&temp, sizeof(unsigned long long) * n * q_amount);

	//generate mu parameters for barrett
	for (int i = 0; i < q_amount; i++)
	{
		unsigned int bit_length = q_bit_lengths[i];
		uint128_t mu1 = uint128_t::exp2(2 * bit_length);
		mu1 = mu1 / q_array[i];
		unsigned long long mu = mu1.low;

		mu_array.push_back(mu);
	}

	unsigned long long* mu_array_p = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		mu_array_p[i] = mu_array[i];
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(mu_cons), mu_array_p, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice, 0);

	//allocate memory for powers of psi root and psi inverse root
	//and fill those arrays
	unsigned long long** psi_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
	unsigned long long** psiinv_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		psi_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
		psiinv_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

		fillTablePsi128(psi_roots[i], q_array[i], psiinv_roots[i], psi_table[i], psiinv_table[i], n);
	}

	//allocate memory for powers of psi root and psi inverse root on device
	//and copy their values from host to device
	unsigned long long* psi_table_device = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * n);
	unsigned long long* psiinv_table_device = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * n);
	hipMalloc(&psi_table_device, sizeof(unsigned long long) * n * q_amount);
	hipMalloc(&psiinv_table_device, sizeof(unsigned long long) * n * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		hipMemcpy(psi_table_device + i * n, psi_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
		hipMemcpy(psiinv_table_device + i * n, psiinv_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
	}

	unsigned long long* c;
	hipMalloc(&c, sizeof(unsigned long long) * n * q_amount * 2);

	unsigned long long** u = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
	for (int i = 0; i < q_amount; i++)
	{
		hipMalloc(&u[i], sizeof(unsigned long long) * n);
	}

	unsigned long long* e;
	hipMalloc(&e, sizeof(unsigned long long) * n * q_amount * 2);

	unsigned long long* m_poly = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

	randomArray128(m_poly, n, t);

	unsigned long long* m_poly_device;
	hipMalloc(&m_poly_device, n * sizeof(unsigned long long));
	hipMemcpy(m_poly_device, m_poly, n * sizeof(unsigned long long), hipMemcpyHostToDevice);

	q_amount--;  // since last polynomials in RNS will be dropped, and some operations are getting affected from it.

	//generate mu parameter of gamma for barrett
	{
		unsigned int bit_length = output_base_bit_lengths[1];
		uint128_t mu1 = uint128_t::exp2(2 * bit_length);
		mu1 = mu1 / gamma;
		mu_gamma = mu1.low;
	}

	//calculate values of punctured q
	for (int i = 0; i < q_amount; i++)
	{
		uint128_t temp = 1;
		for (int j = 0; j < q_amount; j++)
		{
			if (i == j)
				continue;

			temp = host64x2(temp.low, q_array[j]) % q_array[i];
		}

		punctured_q.push_back(temp.low);
		inv_punctured_q.push_back(modinv128(temp.low, q_array[i]));
		inv_punctured_q_array[i] = inv_punctured_q[i];
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(inv_punctured_q_cons), inv_punctured_q_array, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice);

	//pre-compute the values for base change matrix
	unsigned long long* base_change_matrix = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * 2);
	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < q_amount; j++)
		{
			//base_change_matrix = [t0, t1, t2, 
								  //g0, g1, g2] but flattened xd
			uint128_t temp = 1;
			for (int k = 0; k < q_amount; k++)
			{
				if (j == k)
					continue;
				temp = host64x2(temp.low, q_array[k]) % output_base[i];
			}
			base_change_matrix[i * q_amount + j] = temp.low;
		}
	}

	//allocate memory for base change matrix on device and copy the values to it
	unsigned long long* base_change_matrix_device;
	for (int i = 0; i < q_amount; i++)
	{
		hipMalloc(&base_change_matrix_device, sizeof(unsigned long long) * 2 * q_amount);
		hipMemcpy(base_change_matrix_device, base_change_matrix, sizeof(unsigned long long) * 2 * q_amount, hipMemcpyHostToDevice);
	}

	// KEYGEN
	hipEventRecord(start);
	keygen_rns(in, q_amount + 1, q_array, n, secret_key, public_key, streams, temp, mu_array, q_bit_lengths, psi_table_device, psiinv_table_device);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&keygen, start, stop);

	// ENCRYPTION
	hipEventRecord(start);
	encryption_rns(c, public_key, in, u, e, n, streams, q_array, q_bit_lengths, mu_array,
		inv_q_last_mod_q, psi_table_device, psiinv_table_device, m_poly_device, qi_div_t_rns_array_device, q_array_device, t, q_amount + 1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&enc, start, stop);

	// DECRYPTION
	hipEventRecord(start);
	decryption_rns(c, secret_key, q_array, q_bit_lengths, mu_array, psi_table_device, psiinv_table_device,
		n, q_amount, inv_punctured_q, base_change_matrix_device, t, gamma, mu_gamma, output_base, output_base_bit_lengths,
		neg_inv_qs_mult_mod_t_gamma, gamma_div_2, prod_t_gamma_mod_q);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dec, start, stop);

	unsigned long long* decrypted = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
	hipMemcpy(decrypted, c + n * (q_amount - 1), n * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	bool correct = 1;

	for (int i = 0; i < n; i++)
	{
		if (m_poly[i] != decrypted[i])
		{
			correct = 0;
			break;
		}
	}

	cout << "n: " << n << endl;
	cout << "# of qs: " << q_amount + 1 << endl << endl;

	cout << "Time taken for key generation: " << keygen << " millisec." << endl;
	cout << "Time taken for encryption: " << enc << " millisec." << endl;
	cout << "Time taken for decryption: " << dec << " millisec." << endl << endl;

	if (correct)
		cout << "Computations are correct. " << endl;
	else
		cout << "Computations are wrong. " << endl;

	/*cout << "[";
	for (int i = 0; i < 5; i++)
	{
		cout << plain_poly[i] << ", ";
	}
	cout << "]";*/

	return 0;
}