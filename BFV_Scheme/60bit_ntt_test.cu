#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"
#include "ntt_60bit.cuh"
#include "poly_arithmetic.cuh"

#define check 0

int main()
{
    unsigned N = 1024 * 2;

    int size_array = sizeof(unsigned long long) * N;
    int size = sizeof(unsigned long long);

    unsigned long long q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams(q, psi, psiinv, ninv, q_bit, N);

    unsigned long long* psiTable = (unsigned long long*)malloc(size_array);
    unsigned long long* psiinvTable = (unsigned long long*)malloc(size_array);
    fillTablePsi128(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi

    //copy powers of psi and psi inverse tables to device
    unsigned long long* psi_powers, * psiinv_powers;

    hipMalloc(&psi_powers, size_array);
    hipMalloc(&psiinv_powers, size_array);

    hipMemcpy(psi_powers, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(psiinv_powers, psiinvTable, size_array, hipMemcpyHostToDevice);

    cout << "n = " << N << endl;
    cout << "q = " << q << endl;
    cout << "Psi = " << psi << endl;
    cout << "Psi Inverse = " << psiinv << endl;

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    uint128_t mu1 = uint128_t::exp2(bit_length * 2);
    unsigned long long mu = (mu1 / q).low;

    unsigned long long* a;
    hipHostMalloc(&a, sizeof(unsigned long long) * N);
    randomArray128(a, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* b;
    hipHostMalloc(&b, sizeof(unsigned long long) * N);
    randomArray128(b, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* d_a;
    hipMalloc(&d_a, size_array);
    unsigned long long* d_b;
    hipMalloc(&d_b, size_array);

    unsigned long long* refc;
    if (check)
        refc = refPolyMul128(a, b, q, N);

    hipStream_t ntt1, ntt2;
    hipStreamCreate(&ntt1);
    hipStreamCreate(&ntt2);

    hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
    hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);

    forwardNTTdouble(d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers);
    barrett << <N / 256, 256 >> > (d_a, d_b, q, mu, bit_length);
    inverseNTT(d_a, N, ntt1, q, mu, bit_length, psiinv_powers);

    hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, 0);

    hipDeviceSynchronize();

    hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);

    if (check) //check the correctness of results
    {
        for (int i = 0; i < N; i++)
        {
            if (a[i] != refc[i])
            {
                cout << "error" << endl;
                cout << i << "   " << a[i] << "   " << refc[i] << endl;
            }

        }

        free(refc);
    }

    hipHostFree(a); hipHostFree(b);

    return 0;
}


