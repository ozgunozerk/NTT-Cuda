#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"

void decryption_rns(unsigned long long** c0, unsigned long long** c1, unsigned long long** secret_key, 
    vector<unsigned long long>& q, vector<unsigned>& q_bit_lengths, vector<unsigned long long>& mu_array, 
    unsigned long long** psi_table_device, unsigned long long** psiinv_table_device, int n, unsigned q_amount,
    vector<unsigned long long>& inv_punctured_q, unsigned long long** base_change_matrix_device,
    unsigned long long t, unsigned long long gamma, unsigned long long mu_gamma, vector<unsigned long long>& output_base,
    vector<unsigned>& output_base_bit_lengths)
{
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount);
    for (int i = 0; i < q_amount; i++)
        hipStreamCreate(&streams[i]);

    for (int i = 0; i < q_amount; i++)
    {
        half_poly_mul_device(c1[i], secret_key[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device[i], psiinv_table_device[i]);
        // c1 = c1 * sk

        poly_add_device(c1[i], c0[i], n, streams[i], q[i]);
        // c1 = c1 + c0
    }

    //start of fastbconvert array

    // c / punc_q mod q
    for (int i = 0; i < q_amount; i++)
    {
        poly_mul_int(c1[i], inv_punctured_q[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i]);
    }

    fast_convert_array_kernels(c1, c0, t, base_change_matrix_device, q_amount, gamma,
        output_base_bit_lengths[1], mu_gamma, streams[0], streams[1], n);
}

int main()
{
    int n = 1024 * 4;

    vector<unsigned long long> q = { 68719403009, 68719230977, 137438822401 };
    vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };
    vector<unsigned long long> psiinv_roots = { 60243494989, 37410665880, 5716440802 };
    vector<unsigned> q_bit_lengths = { 36, 36, 37 };
    vector<unsigned long long> mu_array = {};
    unsigned q_amount = q.size();

    vector<unsigned long long> punctured_q = {};
    vector<unsigned long long> inv_punctured_q = { 4548496129, 45637159345, 37067270992 };

    unsigned long long t = 1024;
    unsigned long long gamma = 2305843009213683713;

    vector<unsigned long long> output_base = { t, gamma };
    vector<unsigned> output_base_bit_lengths = { 10, 61 };
    unsigned long long mu_gamma;

    //generate mu parameter for barrett
    for (int i = 0; i < q_amount; i++)
    {
        unsigned int bit_length = q_bit_lengths[i];
        uint128_t mu1 = uint128_t::exp2(2 * bit_length);
        mu1 = mu1 / q[i];
        unsigned long long mu = mu1.low;

        mu_array.push_back(mu);
    }

    //generate mu parameter of gamma for barrett
    {
    unsigned int bit_length = output_base_bit_lengths[1];
    uint128_t mu1 = uint128_t::exp2(2 * bit_length);
    mu1 = mu1 / gamma;
    unsigned long long mu_gamma = mu1.low; 
    }

    //allocate memory for powers of psi root and psi inverse root
    //and fill those arrays
    unsigned long long** psi_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        psi_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
        psiinv_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

        fillTablePsi128(psi_roots[i], q[i], psiinv_roots[i], psi_table[i], psiinv_table[i], n);
    }

    //allocate memory for powers of psi root and psi inverse root on device
    //and copy their values from host to device
    unsigned long long** psi_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&psi_table_device[i], sizeof(unsigned long long) * n);
        hipMalloc(&psiinv_table_device[i], sizeof(unsigned long long) * n);

        hipMemcpy(psi_table_device[i], psi_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
        hipMemcpy(psiinv_table_device[i], psiinv_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
    }

    //calculate values of punctured q
    for (int i = 0; i < q_amount; i++)
    {
        uint128_t temp = 1;
        for (int j = 0; j < q_amount; j++)
        {
            if (i == j)
                continue;

            temp = host64x2(temp.low, q[j]) % q[i];
        }

        punctured_q.push_back(temp.low);
    }

    //pre-compute the values for base change matrix
    unsigned long long** base_change_matrix = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        base_change_matrix[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * 2);
    }
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            base_change_matrix[i][j] = punctured_q[j] % output_base[i];
        }
    }

    //allocate memory for base change matrix on device and copy the values to it
    unsigned long long** base_change_matrix_device = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&base_change_matrix_device[i], sizeof(unsigned long long) * 2);
        hipMemcpy(base_change_matrix_device[i], base_change_matrix[i], sizeof(unsigned long long) * 2, hipMemcpyHostToDevice);
    }

    unsigned long long** secret_key = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** c0 = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** c1 = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);

    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&secret_key[i], sizeof(unsigned long long) * n);
        hipMalloc(&c0[i], sizeof(unsigned long long) * n);
        hipMalloc(&c1[i], sizeof(unsigned long long) * n);
    }

    decryption_rns(c0, c1, secret_key, q, q_bit_lengths, mu_array, psi_table_device, psiinv_table_device, 
        n, q_amount, inv_punctured_q, base_change_matrix_device, t, gamma, mu_gamma, output_base, output_base_bit_lengths);
}