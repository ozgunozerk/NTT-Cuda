#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"
#include "distributions.cuh"

#define check 0

int main2()
{
    int n = 1024 * 4;

    vector<unsigned long long> q = { 68719403009, 68719230977, 137438822401 };
    vector<unsigned long long> psi_roots = { 24250113, 29008497, 8625844 };
    vector<unsigned> q_bit_lengths = { 36, 36, 37 };
    unsigned q_amount = q.size();

    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount);
    for (int i = 0; i < q_amount; i++)
        hipStreamCreate(&streams[i]);
    
    unsigned char* in;
    hipMalloc(&in, (sizeof(char) + sizeof(unsigned) + sizeof(unsigned long long)) * q_amount * n);

    unsigned long long** secret_key = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&secret_key[i], sizeof(unsigned long long) * n);
    }
    
    unsigned long long*** public_key = (unsigned long long***)malloc(sizeof(unsigned long long**) * 2);
    public_key[0] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    public_key[1] = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            hipMalloc(&public_key[i][j], sizeof(unsigned long long) * n);
        }
    }
    unsigned long long** temp = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&temp[i], sizeof(unsigned long long) * q_amount * n);
    }

    generate_random(in, (sizeof(char) + sizeof(unsigned) + sizeof(unsigned long long)) * q_amount * n, streams[0]);

    for (int i = 0; i < q_amount; i++)
    {
        ternary_dist(in + i * n, secret_key[i], n, streams[i], q[i]);
    }

    unsigned long long* output;
    hipHostMalloc(&output, sizeof(unsigned long long) * n);
    hipMemcpyAsync(output, secret_key[2], sizeof(unsigned long long) * n, hipMemcpyDeviceToHost, streams[2]);

    hipDeviceSynchronize();

    /*int c = 0, v = 0, b = 0;
    for (int i = 0; i < n; i++)
    {
        if (output[i] == (q[2] - 1))
            c++;
        else if (output[i] == 0)
            v++;
        else if (output[i] == 1)
            b++;
    }

    printf("%d, %d, %d\n", c, v, b);*/


    return 0;
}

int main3()
{
    //unsigned N = atoi(argv[1]);
    unsigned N = 1024 * 8;

    int size_array = sizeof(unsigned long long) * N;
    int size = sizeof(unsigned long long);

    unsigned long long q, psi, psiinv, ninv;
    unsigned int q_bit;

    getParams(q, psi, psiinv, ninv, q_bit, N);

    unsigned long long* psiTable = (unsigned long long*)malloc(size_array);
    unsigned long long* psiinvTable = (unsigned long long*)malloc(size_array);
    fillTablePsi128(psi, q, psiinv, psiTable, psiinvTable, N); //gel psi psi

    //copy powers of psi and psi inverse tables to device
    unsigned long long* psi_powers, * psiinv_powers;

    hipMalloc(&psi_powers, size_array);
    hipMalloc(&psiinv_powers, size_array);

    hipMemcpy(psi_powers, psiTable, size_array, hipMemcpyHostToDevice);
    hipMemcpy(psiinv_powers, psiinvTable, size_array, hipMemcpyHostToDevice);

    cout << "n = " << N << endl;
    cout << "q = " << q << endl;
    cout << "Psi = " << psi << endl;
    cout << "Psi Inverse = " << psiinv << endl;

    //generate parameters for barrett
    unsigned int bit_length = q_bit;
    uint128_t mu1 = uint128_t::exp2(2 * bit_length);
    mu1 = mu1 / q;

    unsigned long long mu = mu1.low;

    unsigned long long* a;
    hipHostMalloc(&a, sizeof(unsigned long long) * N);
    randomArray128(a, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* b;
    hipHostMalloc(&b, sizeof(unsigned long long) * N);
    randomArray128(b, N, q); //fill array with random numbers between 0 and q - 1

    unsigned long long* d_a;
    hipMalloc(&d_a, size_array);
    unsigned long long* d_b;
    hipMalloc(&d_b, size_array);

    unsigned long long* refc;
    if (check)
        refc = refPolyMul128(a, b, q, N);

    hipStream_t ntt1, ntt2;
    hipStreamCreate(&ntt1);
    hipStreamCreate(&ntt2);

    /*hipMemcpyAsync(d_a, a, size_array, hipMemcpyHostToDevice, ntt1);
    hipMemcpyAsync(d_b, b, size_array, hipMemcpyHostToDevice, ntt2);

    full_poly_mul_device(d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers, psiinv_powers);

    hipMemcpyAsync(a, d_a, size_array, hipMemcpyDeviceToHost, 0);*/

    unsigned long long* result = full_poly_mul(a, b, d_a, d_b, N, ntt1, ntt2, q, mu, bit_length, psi_powers, psiinv_powers);

    hipDeviceSynchronize();

    hipStreamDestroy(ntt1); hipStreamDestroy(ntt2);

    if (check) //check the correctness of results
    {
        for (int i = 0; i < N; i++)
        {
            if (result[i] != refc[i])
            {
                cout << "error" << endl;
                cout << i << "   " << a[i] << "   " << refc[i] << endl;
            }

        }

        free(refc);
    }

    hipHostFree(a); hipHostFree(b);

    return 0;
}

/*int main()
{
    int n = 1024 * 4;

    unsigned long long q = 68719403009;

    unsigned char* in;
    unsigned long long* out;
    unsigned long long* host;

    hipHostMalloc(&host, n * sizeof(unsigned long long));

    hipMalloc(&in, n * sizeof(char)); hipMalloc(&out, n * sizeof(unsigned long long));

    hipStream_t stream; hipStreamCreate(&stream);

    generate_random(in, n, stream, 1);
    uniform_dist((unsigned long long*)in, out, n, stream, q);

    hipMemcpyAsync(host, out, n * sizeof(unsigned long long), hipMemcpyDeviceToHost, stream);

    hipDeviceSynchronize();

    for (int i = 0; i < n; i++)
    {
        printf("%llu\n", host[i]);
    }

    return 0;
}*/


