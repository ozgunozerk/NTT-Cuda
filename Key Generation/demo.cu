#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"

#include "poly_arithmetic.cuh"
#include "distributions.cuh"

#define small_block 128

void print_array(unsigned long long a[], unsigned n)
{
    cout << "[";
    for (int i = 0; i < n; i++)
    {
        cout << a[i] << ", ";
    }
    cout << "]\n";
}

__global__ void divide_and_round_q_last_inplace_add_x2(unsigned long long* c, unsigned n, int q_amount)
{
    unsigned long long last_modulus = q_cons[q_amount - 1];  // get the last q from the array
    unsigned long long half_last_modulus = last_modulus >> 1;  // divide it by 2

    int i = blockIdx.x * small_block + threadIdx.x;

    unsigned long long ra = c[n * (q_amount - 1) + i % n + (n * q_amount) * (i >= n)];
    ra += half_last_modulus;

    if (ra >= last_modulus)
        ra -= last_modulus;

    c[n * (q_amount - 1) + i % n + (n * q_amount) * (i >= n)] = ra;

}

__global__ void divide_and_round_q_last_inplace_loop_xq(unsigned long long* c, unsigned q_amount, unsigned n)
{
    int i = blockIdx.x * small_block + threadIdx.x;
    int i_i = i % n;

    unsigned long long last_modulus = q_cons[q_amount - 1];  // get the last q from the array
    unsigned long long half_last_modulus = last_modulus >> 1;  // divide it by 2

    unsigned index = (i % (n * (q_amount - 1))) / n;

    unsigned long long q = q_cons[index];
    unsigned long long mu = mu_cons[index];
    int q_bit = q_bit_cons[index];
    unsigned long long half_mod = half_last_modulus % q;

    unsigned long long inv_q_last_mod_q = inv_q_last_mod_q_cons[index];

    unsigned second_half = i >= (n * (q_amount - 1));
    unsigned division = (i - n * second_half * (q_amount - 1)) / n;
    unsigned long long* rns_poly_minus1 = c + second_half * (n * q_amount) + n * (q_amount - 1);
    unsigned long long* input_poly = c + second_half * (n * q_amount) + n * division;

    unsigned long long temp_poly_i = rns_poly_minus1[i_i] % q;
    // get the last polynomials respective index with = rns_poly_minus1[i] 
    // get the the base q of the polynomial (one of other polynomials other than the last one) = base_q_i 
    // store the result in a variable = temp_poly_i

    if (temp_poly_i < half_mod)  // mod operation for safe substraction on line 139
        temp_poly_i += q;

    temp_poly_i -= half_mod; // substract half_modulus from the index of last polynomial

    if (input_poly[i_i] < temp_poly_i)  // now we gonna substract that computed value from other polynomials
        input_poly[i_i] += q;  // so we have to ensure substraction safety (underflow)

    input_poly[i_i] -= temp_poly_i;
    // substract the last_polynomials respective calculated value = temp_poly_i
    // from the respective index of the other polynomial = input_poly[i]

    uint128_t mult;
    mul64(input_poly[i_i], inv_q_last_mod_q, mult);
    // multiply the input_poly[i] with:
    // inverse of last polynomials q
    // to the mod of respective polynomials q
    // which is: inv_q_last_mod_q_i 
    // :)

    singleBarrett(mult, q, mu, q_bit);
    // we might have fucked up, so apply mod again

    input_poly[i_i] = mult.low;  // store the result in the given input_polynomial
}

__global__ void convert_ternary_gaussian_x2(unsigned char* in, unsigned long long* out_t1, unsigned long long* out_e1, unsigned n, int q_amount)
{
    int i = blockIdx.x * convertBlockSize + threadIdx.x;

    //ternary

    float d = (float)in[i % n];

    d /= (256.0f / 3);

    if (d >= 2)
        out_t1[i] = 1;
    else if (d >= 1)
        out_t1[i] = 0;
    else
        out_t1[i] = q_cons[i / n] - 1;

    if (d >= 2)
        out_t1[i + n * q_amount] = 1;
    else if (d >= 1)
        out_t1[i + n * q_amount] = 0;
    else
        out_t1[i + n * q_amount] = q_cons[i / n] - 1;

    // gaussian

    unsigned* in_u = (unsigned*)(in + n);
    d = in_u[i % n];

    d /= 4294967295;

    if (d == 0)
        d += 1.192092896e-07F;
    else if (d == 1)
        d -= 1.192092896e-07F;

    d = normcdfinvf(d);

    d = d * (float)dstdev + dmean;

    if (d > 19.2)
    {
        d = 19.2;
    }
    else if (d < -19.2)
    {
        d = -19.2;
    }

    int dd = (int)d;

    if (dd < 0)
        out_e1[i] = q_cons[i / n] + dd;
    else
        out_e1[i] = dd;

    in_u = (unsigned*)(in + n * 5);
    d = in_u[i % n];

    d /= 4294967295;

    if (d == 0)
        d += 1.192092896e-07F;
    else if (d == 1)
        d -= 1.192092896e-07F;

    d = normcdfinvf(d);

    d = d * (float)dstdev + dmean;

    if (d > 19.2)
    {
        d = 19.2;
    }
    else if (d < -19.2)
    {
        d = -19.2;
    }

    dd = (int)d;

    if (dd < 0)
        out_e1[i + n * q_amount] = q_cons[i / n] + dd;
    else
        out_e1[i + n * q_amount] = dd;

}

__global__ void poly_add_xq(unsigned long long* c, unsigned long long* e, unsigned n, int q_amount)
{
    int i = blockIdx.x * small_block + threadIdx.x;

    unsigned long long ra = c[i] + e[i];

    if (ra > q_cons[i / n])
        ra -= q_cons[i / n];

    c[i] = ra;

    ra = c[n * q_amount + i] + e[n * q_amount + i]; //optimise later

    if (ra > q_cons[i / n])
        ra -= q_cons[i / n];

    c[n * q_amount + i] = ra;
}

void encryption_rns(unsigned long long* c, unsigned long long* public_key, unsigned char* in, unsigned long long** u, unsigned long long* e, unsigned N,
    hipStream_t streams[], unsigned long long* q, vector<unsigned> q_bit_lengths,
    vector<unsigned long long> mu_array, vector<unsigned long long> inv_q_last_mod_q, unsigned long long* psi_table_device, unsigned long long* psiinv_table_device,
    unsigned long long* m_poly_device, unsigned long long* qi_div_t_rns_array_device, unsigned long long* q_array_device, unsigned t, int q_amount)
{
    generate_random_default(in, sizeof(char) * N + sizeof(unsigned) * N * 2);  // default is for default stream: this is for synchronization issues
    // otherwise ternary distributions may run before this function, which is UNACCEPTABLE

    /*for (int i = 0; i < q_amount; i++)
    {
        ternary_dist(in, c + i * N, N, streams[i], q[i]);  // generate ternary dist poly directly into c0 and c1. c0 = c1, 
        ternary_dist(in, c + i * N + q_amount * N, N, streams[i], q[i]);  // its represented by 'u' 
        // for ease of operations and memory allocation, we have generated 2 of them (identical), since we override some stuff in polynomial multiplication.
    }

    for (int i = 0; i < q_amount; i++)
    {
        gaussian_dist((unsigned*)(in + N), e + i * N, N, streams[i], q[i]);  // this is again for generation ternary distribution, although it's name is gaussian
        // e0

        gaussian_dist((unsigned*)(in + N + N * 4), e + i * N + N * q_amount, N, streams[i], q[i]);  // i was joking this is for gaussian
        // e1
    }*/

    int convert_block_amount = q_amount * N / convertBlockSize;
    convert_ternary_gaussian_x2 << <convert_block_amount, convertBlockSize, 0, 0 >> > (in, c, e, N, q_amount);

    /*for (int i = 0; i < q_amount; i++)
    {
        // multiply each public key with 'u'(c0 and c1). Remember that c0 and c1 are identical
        half_poly_mul_device(c + i * N, public_key + i * N, N, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device + i * N, psiinv_table_device + i * N);
        half_poly_mul_device(c + i * N + q_amount * N, public_key + i * N + q_amount * N, N, streams[i + q_amount], q[i], mu_array[i], q_bit_lengths[i], psi_table_device + i * N, psiinv_table_device + i * N);
    }*/

    forwardNTT_batch(c, N, psi_table_device, q_amount * 2, q_amount);
    dim3 barrett_dim(N / 256, q_amount * 2);
    barrett_batch<<< barrett_dim, 256, 0, 0>>>(c, public_key, N, q_amount);
    inverseNTT_batch(c, N, psiinv_table_device, q_amount * 2, q_amount);

    /*for (int i = 0; i < q_amount; i++)
    {
        poly_add_device(c + i * N, e + i * N, N, streams[i], q[i]);  // add e0 to publickey[0]
        poly_add_device(c + i * N + q_amount * N, e + i * N + N * q_amount, N, streams[i + q_amount], q[i]);  // add e1 to publickey[1]
    }*/

    poly_add_xq << <N * q_amount / small_block, small_block, 0, 0 >> > (c, e, N, q_amount);

    divide_and_round_q_last_inplace_add_x2 << <N * 2 / small_block, small_block, 0, 0 >> > (c, N, q_amount);

    /*divide_and_round_q_last_inplace(c, N, streams, q, q_bit_lengths, mu_array, inv_q_last_mod_q, q_amount);  // do that complicated stuff for each public key
    divide_and_round_q_last_inplace(c + q_amount * N, N, streams, q, q_bit_lengths, mu_array, inv_q_last_mod_q, q_amount);*/

    divide_and_round_q_last_inplace_loop_xq << <N * 2 * (q_amount - 1) / small_block, small_block, 0, 0 >> > (c, q_amount, N);

    weird_m_stuff << <N / 256, 256, 0, 0 >> > (m_poly_device, c, t, qi_div_t_rns_array_device, q_array_device, q_amount, N);  // look at the comments in the function
}

void decryption_rns(unsigned long long* c, unsigned long long** secret_key,
    unsigned long long* q, vector<unsigned>& q_bit_lengths, vector<unsigned long long>& mu_array,
    unsigned long long* psi_table_device, unsigned long long* psiinv_table_device, int n, unsigned q_amount,
    vector<unsigned long long>& inv_punctured_q, unsigned long long* base_change_matrix_device,
    unsigned long long t, unsigned long long gamma, unsigned long long mu_gamma, vector<unsigned long long>& output_base,
    vector<unsigned>& output_base_bit_lengths, vector<unsigned long long>& neg_inv_q_mod_t_gamma,
    unsigned long long gamma_div_2, vector<unsigned long long> prod_t_gamma_mod_q) // hehehehe
{
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount);
    for (int i = 0; i < q_amount; i++)
        hipStreamCreate(&streams[i]);

    for (int i = 0; i < q_amount; i++)
    {
        half_poly_mul_device(c + i * n + (q_amount + 1) * n, secret_key[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device + i * n, psiinv_table_device + i * n);
        // c1 = c1 * sk

        poly_add_device(c + i * n + (q_amount + 1) * n, c + i * n, n, streams[i], q[i]);
        // c1 = c1 + c0
    }

    for (int i = 0; i < q_amount; i++)
    {
        poly_mul_int(c + i * n + (q_amount + 1) * n, prod_t_gamma_mod_q[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i]);
        // c1 = c1 * prod_t_gamma_mod_q
    }

    // start of fast convert array

    // c1 / punc_q mod q
    for (int i = 0; i < q_amount; i++)
    {
        poly_mul_int(c + i * n + (q_amount + 1) * n, inv_punctured_q[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i]);
    }

    //hipStreamSynchronize(streams[q_amount - 1]);

    // multiply coeff[k] with base change matrix, add them together and split into 2 poly
    fast_convert_array_kernels(c + (q_amount + 1) * n, c, t, base_change_matrix_device, q_amount, gamma,
        output_base_bit_lengths[1], mu_gamma, streams[0], streams[1], n);

    // end of fast convert array

    // multiply polies by neg_inv_q_mod_t_gamma
    poly_mul_int_t(c, neg_inv_q_mod_t_gamma[0], n, streams[0], t);
    poly_mul_int(c + n, neg_inv_q_mod_t_gamma[1], n, streams[1], gamma, mu_gamma, output_base_bit_lengths[1]);

    //round
    dec_round(c, c + n * (q_amount - 1), t, gamma, gamma_div_2, n, streams[1]);
}

void keygen_rns(unsigned char in[], int q_amount, unsigned long long* q, unsigned n, unsigned long long** secret_key, unsigned long long* public_key,
    hipStream_t* streams, unsigned long long** temp, vector<unsigned long long> mu_array, vector<unsigned> q_bit_lengths, 
    unsigned long long* psi_table_device, unsigned long long* psiinv_table_device)
{
    generate_random_default(in, (sizeof(char) + sizeof(unsigned long long)) * q_amount * n + sizeof(unsigned) * n);

    // convert random bytes to ternary distribution
    // use same byte sequence for each element of the secret key
    for (int i = 0; i < q_amount; i++)
    {
        ternary_dist(in, secret_key[i], n, streams[i], q[i]);
    }

    // convert random bytes to uniform distribution
    // use different byte sequences for each q
    for (int i = 0; i < q_amount; i++)
    {
        uniform_dist((unsigned long long*)(in + n + i * n * sizeof(unsigned long long)), public_key + i * n + q_amount * n, n, streams[i], q[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        gaussian_dist((unsigned*)(in + n + q_amount * n * sizeof(unsigned long long)), temp[i], n, streams[i], q[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        forwardNTT(secret_key[i], n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device + i * n);
    }

    for (int i = 0; i < q_amount; i++)
    {
        hipMemcpyAsync(public_key + i * n, public_key + i * n + q_amount * n, sizeof(unsigned long long) * n, hipMemcpyDeviceToDevice, streams[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        barrett << <n / 256, 256, 0, streams[i] >> > (public_key + i * n, secret_key[i], q[i], mu_array[i], q_bit_lengths[i]);
        inverseNTT(public_key + i * n, n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psiinv_table_device + i * n);
        poly_add_device(public_key + i * n, temp[i], n, streams[i], q[i]);
        poly_negate_device(public_key + i * n, n, streams[i], q[i]);
    }

    for (int i = 0; i < q_amount; i++)
    {
        forwardNTT(public_key + i * n, n, streams[i], q[i], mu_array[i], q_bit_lengths[i], psi_table_device + i * n);
    }
}

int main()
{
    //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    int n = 1024 * 16;

    unsigned long long t = 1024;  // mathematical stuff that is beyond our comprehension
    vector<unsigned long long> q = { 281474976546817, 281474976317441, 281474975662081, 562949952798721, 562949952700417, 562949952274433, 562949951979521, 562949951881217, 36028797017456641 };
    unsigned long long q_array[] = { 281474976546817, 281474976317441, 281474975662081, 562949952798721, 562949952700417, 562949952274433, 562949951979521, 562949951881217, 36028797017456641 };
    vector<unsigned long long> psi_roots = { 23720796222, 21741529212, 13412349256, 1196930505, 31695302805, 6575376104, 394024808, 45092463253, 1155186985540 };

    /*vector<unsigned long long> q = { 36028797017456641, 36028797014704129, 36028797014573057, 36028797014376449, 36028797013327873, 36028797013000193, 36028797012606977, 36028797010444289, 36028797009985537, 36028797005856769, 36028797005529089, 36028797005135873, 36028797003694081, 36028797003563009, 36028797001138177 };
    unsigned long long q_array[] = { 36028797017456641, 36028797014704129, 36028797014573057, 36028797014376449, 36028797013327873, 36028797013000193, 36028797012606977, 36028797010444289, 36028797009985537, 36028797005856769, 36028797005529089, 36028797005135873, 36028797003694081, 36028797003563009, 36028797001138177 };
    vector<unsigned long long> psi_roots = { 1155186985540, 631260524634, 1526647220035, 455957817523, 1650884166641, 10316746886, 768741990072, 3911086673862, 5947090524825, 47595902954, 2691682578057, 3903338373, 235185854118, 1769787302793, 3151164484090 };*/

    /*unsigned long long q_array[] = { 274877562881, 274877202433, 274877153281 };
    vector<unsigned long long> psi_roots = { 71485851, 33872056, 22399294 };*/

    /*unsigned long long q_array[] = { 8796092858369, 8796092792833, 17592186028033, 17592185438209 };
    vector<unsigned long long> psi_roots = { 1734247217, 304486499, 331339694, 9366611238 };*/

    /*vector<unsigned long long> q = { 1125899904679937, 1125899903991809, 1125899903827969, 1125899903795201, 1125899903500289 };
    unsigned long long q_array[] = { 1125899904679937, 1125899903991809, 1125899903827969, 1125899903795201, 1125899903500289 };
    vector<unsigned long long> psi_roots = { 184459094098, 125929543876, 13806300337, 10351677219, 68423600398 };*/

    vector<unsigned long long> psiinv_roots;
    vector<unsigned long long> mu_array = {};
    unsigned q_amount = sizeof(q_array) / sizeof(unsigned long long);
    vector<unsigned> q_bit_lengths;
    unsigned* q_bit_lengths_p = (unsigned*)malloc(sizeof(unsigned) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        q_bit_lengths.push_back(log2(q_array[i]) + 1);
        q_bit_lengths_p[i] = q_bit_lengths[i];
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(q_bit_cons), q_bit_lengths_p, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice, 0);
    vector<unsigned long long> inv_q_last_mod_q;
    unsigned long long* inv_q_last_mod_q_p = (unsigned long long*)malloc(sizeof(unsigned long long) * (q_amount - 1));
    for (int i = 0; i < q_amount - 1; i++)
    {
        inv_q_last_mod_q.push_back(modinv128(q_array[q_amount - 1] % q_array[i], q_array[i]));
        inv_q_last_mod_q_p[i] = inv_q_last_mod_q[i];
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(inv_q_last_mod_q_cons), inv_q_last_mod_q_p, sizeof(unsigned long long) * (q_amount - 1), 0, hipMemcpyHostToDevice, 0);
    unsigned long long q_mod_t = 1;
    //unsigned long long qi_div_t_rns_array[] = { 67108792, 67108624, 134217600 };
    unsigned long long* qi_div_t_rns_array = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        qi_div_t_rns_array[i] = q_array[i] / t;
    }
    vector<unsigned long long> punctured_q = {};
    //vector<unsigned long long> inv_punctured_q = { 26179219651, 42540076863 };
    vector<unsigned long long> inv_punctured_q;
    unsigned long long gamma = 2305843009213683713;
    unsigned long long gamma_div_2 = gamma >> 1;

    for (int i = 0; i < q_amount; i++)
        psiinv_roots.push_back(modinv128(psi_roots[i], q_array[i]));

    vector<unsigned long long> output_base = { t, gamma };
    vector<unsigned> output_base_bit_lengths = { 10, 61 };
    unsigned long long mu_gamma;

    unsigned long long mult_t = 1, mult_g = 1;
    for (int i = 0; i < q_amount - 1; i++)
    {
        mult_t = (host64x2(mult_t, q_array[i]) % t).low;
        mult_g = (host64x2(mult_g, q_array[i]) % gamma).low;
    }
    mult_t = t - modinv128(mult_t, t);
    mult_g = gamma - modinv128(mult_g, gamma);

    vector<unsigned long long> neg_inv_qs_mult_mod_t_gamma = { mult_t, mult_g }; // all qs multiplied mod t and gamma then taken inverse then negated

    //vector<unsigned long long> prod_t_gamma_mod_q = { 37067052033, 64547873793 };
    vector<unsigned long long> prod_t_gamma_mod_q;
    uint128_t prod_t_gamma = host64x2(t, gamma);
    for (int i = 0; i < q_amount; i++)
    {
        prod_t_gamma_mod_q.push_back((prod_t_gamma % q_array[i]).low);
    }

    hipMemcpyToSymbolAsync(HIP_SYMBOL(q_cons), q_array, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice);

    // run operations on different q's with different streams
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * q_amount * 2);
    for (int i = 0; i < q_amount * 2; i++)
        hipStreamCreate(&streams[i]);

    unsigned long long* q_array_device;
    hipMalloc(&q_array_device, q_amount * sizeof(unsigned long long));
    hipMemcpy(q_array_device, q_array, q_amount * sizeof(unsigned long long), hipMemcpyHostToDevice);

    unsigned long long* qi_div_t_rns_array_device;
    hipMalloc(&qi_div_t_rns_array_device, q_amount * sizeof(unsigned long long));
    hipMemcpy(qi_div_t_rns_array_device, qi_div_t_rns_array, q_amount * sizeof(unsigned long long), hipMemcpyHostToDevice);

    unsigned char* in;
    hipMalloc(&in, (sizeof(char) + sizeof(unsigned long long)) * q_amount * n + sizeof(unsigned) * n);

    unsigned long long** secret_key = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&secret_key[i], sizeof(unsigned long long) * n);
    }

    // we always have 2 public keys
    unsigned long long* public_key;
    hipMalloc(&public_key, sizeof(unsigned long long) * 2 * n * q_amount);

    // a temp array to store gaussian distribution values (e)
    unsigned long long** temp = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&temp[i], sizeof(unsigned long long) * n);
    }

    //generate mu parameters for barrett
    for (int i = 0; i < q_amount; i++)
    {
        unsigned int bit_length = q_bit_lengths[i];
        uint128_t mu1 = uint128_t::exp2(2 * bit_length);
        mu1 = mu1 / q_array[i];
        unsigned long long mu = mu1.low;

        mu_array.push_back(mu);
    }

    unsigned long long* mu_array_p = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        mu_array_p[i] = mu_array[i];
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(mu_cons), mu_array_p, sizeof(unsigned long long) * q_amount, 0, hipMemcpyHostToDevice, 0);

    //allocate memory for powers of psi root and psi inverse root
    //and fill those arrays
    unsigned long long** psi_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    unsigned long long** psiinv_table = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        psi_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
        psiinv_table[i] = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

        fillTablePsi128(psi_roots[i], q_array[i], psiinv_roots[i], psi_table[i], psiinv_table[i], n);
    }

    //allocate memory for powers of psi root and psi inverse root on device
    //and copy their values from host to device
    unsigned long long* psi_table_device = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * n);
    unsigned long long* psiinv_table_device = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * n);
    hipMalloc(&psi_table_device, sizeof(unsigned long long) * n * q_amount);
    hipMalloc(&psiinv_table_device, sizeof(unsigned long long) * n * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMemcpy(psi_table_device + i * n, psi_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
        hipMemcpy(psiinv_table_device + i * n, psiinv_table[i], sizeof(unsigned long long) * n, hipMemcpyHostToDevice);
    }

    unsigned long long* c;
    hipMalloc(&c, sizeof(unsigned long long) * n * q_amount * 2);

    unsigned long long** u = (unsigned long long**)malloc(sizeof(unsigned long long*) * q_amount);
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&u[i], sizeof(unsigned long long) * n);
    }

    unsigned long long* e;
    hipMalloc(&e, sizeof(unsigned long long) * n * q_amount * 2);

    unsigned long long* m_poly = (unsigned long long*)malloc(sizeof(unsigned long long) * n);
    for (int i = 0; i < n; i++)
    {
        m_poly[i] = 0;
    }

    randomArray128(m_poly, n, t);
    //m_poly[0] = 1; m_poly[1] = 1; m_poly[2] = 1; m_poly[3] = 1;
    unsigned long long* m_poly_device;
    hipMalloc(&m_poly_device, n * sizeof(unsigned long long));
    hipMemcpy(m_poly_device, m_poly, n * sizeof(unsigned long long), hipMemcpyHostToDevice);

    q_amount--;

    //generate mu parameter of gamma for barrett
    {
        unsigned int bit_length = output_base_bit_lengths[1];
        uint128_t mu1 = uint128_t::exp2(2 * bit_length);
        mu1 = mu1 / gamma;
        mu_gamma = mu1.low;
    }

    //calculate values of punctured q
    for (int i = 0; i < q_amount; i++)
    {
        uint128_t temp = 1;
        for (int j = 0; j < q_amount; j++)
        {
            if (i == j)
                continue;

            temp = host64x2(temp.low, q_array[j]) % q_array[i];
        }

        punctured_q.push_back(temp.low);
        inv_punctured_q.push_back(modinv128(temp.low, q_array[i]));
    }

    //pre-compute the values for base change matrix
    unsigned long long* base_change_matrix = (unsigned long long*)malloc(sizeof(unsigned long long) * q_amount * 2);
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < q_amount; j++)
        {
            //base_change_matrix = [t0, t1, t2, 
                                  //g0, g1, g2] but flattened xd
            uint128_t temp = 1;
            for (int k = 0; k < q_amount; k++)
            {
                if (j == k)
                    continue;
                temp = host64x2(temp.low, q_array[k]) % output_base[i];
            }
            base_change_matrix[i * q_amount + j] = temp.low;
        }
    }

    //allocate memory for base change matrix on device and copy the values to it
    unsigned long long* base_change_matrix_device;
    for (int i = 0; i < q_amount; i++)
    {
        hipMalloc(&base_change_matrix_device, sizeof(unsigned long long) * 2 * q_amount);
        hipMemcpy(base_change_matrix_device, base_change_matrix, sizeof(unsigned long long) * 2 * q_amount, hipMemcpyHostToDevice);
    }

    keygen_rns(in, q_amount + 1, q_array, n, secret_key, public_key, streams, temp, mu_array, q_bit_lengths, psi_table_device, psiinv_table_device);

    encryption_rns(c, public_key, in, u, e, n, streams, q_array, q_bit_lengths, mu_array,
        inv_q_last_mod_q, psi_table_device, psiinv_table_device, m_poly_device, qi_div_t_rns_array_device, q_array_device, t, q_amount + 1);

    decryption_rns(c, secret_key, q_array, q_bit_lengths, mu_array, psi_table_device, psiinv_table_device,
        n, q_amount, inv_punctured_q, base_change_matrix_device, t, gamma, mu_gamma, output_base, output_base_bit_lengths,
        neg_inv_qs_mult_mod_t_gamma, gamma_div_2, prod_t_gamma_mod_q);

    unsigned long long* plain_poly = (unsigned long long*)malloc(sizeof(unsigned long long) * n);

    hipMemcpy(plain_poly, c + n * (q_amount - 1), n * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < n; i++)
    {
        if (m_poly[i] != plain_poly[i])
            cout << "Error " << i << endl;
    }

    /*cout << "[";
    for (int i = 0; i < 5; i++)
    {
        cout << plain_poly[i] << ", ";
    }
    cout << "]";*/

    return 0;
}